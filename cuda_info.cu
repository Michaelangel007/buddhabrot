/*
    nvcc cuda_info.cu -o bin/cuda_info
*/

#include <stdio.h>
#include <hip/hip_runtime.h>

/*
   GeForce GTX Titan @ 928 MHz
      SM: 14 * 192 sm/core = 2688 cores
      384-bit @ 3004 MHz = 288 GB/s

   GeForce GT 750M @ 925 MHz
      SM: 2 * 192 sm/core = 384 cores
      128-bit @ 2508 MHz = 80 GB/s

   GeForce GT 330M @ 1100 MHz
      SM: 6 * 8 sm/core = 48 cores
      128-bit @ 790 MHz = 25 GB/s
*/

int CudaGetCores( int major, int minor )
{
    int cores[] = {
          8,  8, 8, 8, 0,   0, // 1.0  1.1  1.2  1.3
         32, 48, 0, 0, 0,   0, // 2.0  2.1
        192,  0, 0, 0, 0, 192, // 3.0                 3.5
        256,  0, 0, 0, 0,   0
    };

    return cores[ 6*(major-1) + minor ];
}

// hipDeviceProp_t
// Reference: http://developer.download.nvidia.com/compute/cuda/4_1/rel/toolkit/docs/online/group__CUDART__DEVICE_g5aa4f47938af8276f08074d09b7d520c.html
int main()
{
    int nDevices; 
    hipError_t error = hipGetDeviceCount( &nDevices );
    if( error != hipSuccess )
        return printf( "ERROR: Couldn't find any CUDA devices!\n" );

    for( int iDevice = 0; iDevice < nDevices; iDevice++ )
    {
        hipDeviceProp_t prop;

        hipGetDeviceProperties( &prop, iDevice );

        printf( "\nGPU #%d: \'%s\' @ %d MHz\n", (iDevice+1), prop.name, (prop.clockRate/1000) );
        printf( "   Compute: %d.%d\n", prop.major, prop.minor );
        printf( "   Multi Processors: %d * %d Cores/SM = %d Cores\n"
           , prop.multiProcessorCount 
           , CudaGetCores( prop.major, prop.minor )
           , prop.multiProcessorCount * CudaGetCores( prop.major, prop.minor )
        );

        printf( "\n=== Memory ===\n" );
        printf( "   Total Memory : %lu MB (%lu bytes)\n", (prop.totalGlobalMem/1024)/1024, (size_t)prop.totalGlobalMem );
        printf( "   Bus Width    : %u-bit @ %d MHz ==> ", prop. memoryBusWidth, prop.memoryClockRate/1000 );
        printf( "   Max Bandwidth: %u GB/s\n"           , (prop.memoryClockRate/1000 * ((prop. memoryBusWidth/8)*2))/1000 ); // DDR2/3/4/5 = *2
        printf( "   Const memory : %lu (bytes)\n"       , prop.totalConstMem     );
        printf( "   Memory/Block : %lu\n"               , prop.sharedMemPerBlock );
        printf( "   Unified mem  : %d\n"                , prop.unifiedAddressing );

        printf( "\n=== Threads ===\n" );
        printf( "   Max Threads/SM : %d \n"       , prop.maxThreadsPerMultiProcessor );
        printf( "   Threads / Block: %d\n"        , prop.maxThreadsPerBlock );
        printf( "   Max Thread Size: %d, %d, %d\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2] );
        printf( "   Max Grid size  : %u, %u, %u\n", prop.maxGridSize  [0], prop.maxGridSize  [1], prop.maxGridSize  [2] );
        printf( "   Registers/Block: %d\n"        , prop.regsPerBlock );

        printf( "\n=== Texture ===\n" );
        printf( "   Texture Size 1D: %d          \n", prop.maxTexture1D                                                );
        printf( "   Texture Size 2D: %d x %d     \n", prop.maxTexture2D[0], prop.maxTexture2D[1]                       );
        printf( "   Texture Size 3D: %d x %d x %d\n", prop.maxTexture3D[0], prop.maxTexture3D[1], prop.maxTexture3D[2] );

        printf( "\n" );
    }

    return 0;
}

